#include "hip/hip_runtime.h"
#include "./common.cpp"

#if defined(NDEBUG)
#define CUDA_CHECK_ERROR()  0
#else
#define CUDA_CHECK_ERROR() do {\
    hipError_t e = hipGetLastError(); \
    if(hipSuccess != e) { \
        printf("cuda failure \"%s\" at %s:%d\n",\
        hipGetErrorString(e),__FILE__,__LINE__);\
        exit(1);\
    } \
} while(0)
#endif

__global__ void add_kernel(float* b, const float* a)
{
    int i = threadIdx.x;
    b[i] = a[i] + 1.0f;
}

int main()
{
    int SIZE = 8;
    const float a[SIZE] = {0.,1.,2.,3.,4.,5.,6.,7.};
    float b[SIZE] = {0.};

    printf("a = {%f,%f,%f,%f,%f,%f,%f,%f}\n",a[0],a[1],a[2],a[3],a[4],a[5],a[6],a[7]);
    fflush(stdout);

    float* dev_a = nullptr;
    float* dev_b = nullptr;

    hipMalloc((void**)&dev_a,SIZE*sizeof(float));
    hipMalloc((void**)&dev_b,SIZE*sizeof(float));
    hipMemcpy(dev_a,a,SIZE*sizeof(int),hipMemcpyHostToDevice);

    add_kernel<<<1,SIZE>>>(dev_b,dev_a);
    hipDeviceSynchronize();

    hipMemcpy(b,dev_b,SIZE*sizeof(int),hipMemcpyDeviceToDevice);

    printf("b = {%f,%f,%f,%f,%f,%f,%f,%f}\n",b[0],b[1],b[2],b[3],b[4],b[5],b[6],b[7]);

    CUDA_CHECK_ERROR();

    fflush(stdout);

    hipFree(dev_a);
    hipFree(dev_b);

    return 0;
}