#include "hip/hip_runtime.h"
#include "./common.cpp"
//CUDA_CHECK_ERROR();
unsigned vecSize = 256 * 1024 * 1024;
float saxpy_a = 1.234f;

__global__ void kernelSAXPY(float* z, const float a, const float* x, const float* y, unsigned n)
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
    {
        //z[i] = a * x[i] + y[i];
        z[i] = fmaf(a,x[i],y[i]);
    }
}

int main()
{
    float* vecX = new float[vecSize];
    float* vecY = new float[vecSize];
    float* vecZ = new float[vecSize];
	// set random data
	srand( 0 );
	setNormalizedRandomData( vecX, vecSize );
	setNormalizedRandomData( vecY, vecSize );

    float* dev_vecX = nullptr;
    float* dev_vecY = nullptr;
    float* dev_vecZ = nullptr;

    hipMalloc((void**)&dev_vecX, vecSize * sizeof(float));
    hipMalloc((void**)&dev_vecY, vecSize * sizeof(float));
    hipMalloc((void**)&dev_vecZ, vecSize * sizeof(float));
    CUDA_CHECK_ERROR();

    ELAPSED_TIME_BEGIN(1);
    hipMemcpy(dev_vecX,vecX,vecSize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(dev_vecY,vecY,vecSize*sizeof(float),hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR();

    // CUDA kernel launch
	dim3 dimBlock( 1024, 1, 1 );
	dim3 dimGrid( (vecSize + (dimBlock.x - 1)) / dimBlock.x, 1, 1 );

    ELAPSED_TIME_BEGIN(0);
    kernelSAXPY<<<dimGrid,dimBlock>>>(dev_vecZ,saxpy_a,dev_vecX,dev_vecY,vecSize);
    hipDeviceSynchronize();
    ELAPSED_TIME_END(0);
    CUDA_CHECK_ERROR();

    hipMemcpy(vecZ,dev_vecZ,vecSize*sizeof(float),hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR();
    ELAPSED_TIME_END(1);

    
    hipFree(dev_vecX);
    hipFree(dev_vecY);
    hipFree(dev_vecZ);
    CUDA_CHECK_ERROR();

// check the result
	float sumX = getSum( vecX, vecSize );
	float sumY = getSum( vecY, vecSize );
	float sumZ = getSum( vecZ, vecSize );
	float diff = fabsf( sumZ - (saxpy_a * sumX + sumY) );
	printf("SIZE = %d\n", vecSize);
	printf("a    = %f\n", saxpy_a);
	printf("sumX = %f\n", sumX);
	printf("sumY = %f\n", sumY);
	printf("sumZ = %f\n", sumZ);
	printf("diff(sumZ, a*sumX+sumY) =  %f\n", diff);
	printf("diff(sumZ, a*sumX+sumY)/SIZE =  %f\n", diff / vecSize);
	printVec( "vecX", vecX, vecSize );
	printVec( "vecY", vecY, vecSize );
	printVec( "vecZ", vecZ, vecSize );
	// cleaning
	delete[] vecX;
	delete[] vecY;
	delete[] vecZ;
	// done
	return 0;

    
}