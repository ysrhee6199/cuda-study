#include "hip/hip_runtime.h"
#include "./common.cpp"
const unsigned SIZE = 1024 * 1024; // 1M elements

__global__ void singleKernelVecAdd(float* c, const float* a, const float* b, unsigned n)
{
    unsigned i = threadIdx.x;
    if( i < n) // 배열 크기 보다 쓰레드의 크기가 더 클 수 있음
    {
        c[i] = a[i] + b[i];
    }
}

int main(void){
    
    float* vecA = new float[SIZE];
    float* vecB = new float[SIZE];
    float* vecC = new float[SIZE];

    srand(0);
    setNormalizedRandomData(vecA, SIZE);
    setNormalizedRandomData(vecB, SIZE);

    float* dev_vecA = nullptr;
	float* dev_vecB = nullptr;
	float* dev_vecC = nullptr;

    hipMalloc((void**)&dev_vecA, SIZE*sizeof(float));
    hipMalloc((void**)&dev_vecB,SIZE*sizeof(float));
    hipMalloc((void**)&dev_vecC, SIZE * sizeof(float));
    ELAPSED_TIME_BEGIN(1);
    hipMemcpy(dev_vecA,vecA,SIZE*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(dev_vecB,vecB,SIZE*sizeof(float),hipMemcpyHostToDevice);
    
    ELAPSED_TIME_BEGIN(0);
    singleKernelVecAdd<<<1,SIZE>>>(dev_vecC,dev_vecB,dev_vecA,SIZE);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();
    ELAPSED_TIME_END(0);

    hipMemcpy(vecC,dev_vecC,SIZE*sizeof(float),hipMemcpyDeviceToHost);
    ELAPSED_TIME_END(1);
    hipFree(dev_vecA);
    hipFree(dev_vecB);
    hipFree(dev_vecC);

	CUDA_CHECK_ERROR();
	// check the result
	float sumA = getSum( vecA, SIZE );
	float sumB = getSum( vecB, SIZE );
	float sumC = getSum( vecC, SIZE );
	float diff = fabsf( sumC - (sumA + sumB) );
	printf("SIZE = %d\n", SIZE);
	printf("sumA = %f\n", sumA);
	printf("sumB = %f\n", sumB);
	printf("sumC = %f\n", sumC);
	printf("diff(sumC, sumA+sumB) =  %f\n", diff);
	printf("diff(sumC, sumA+sumB) / SIZE =  %f\n", diff / SIZE);
	printVec( "vecA", vecA, SIZE );
	printVec( "vecB", vecB, SIZE );
	printVec( "vecC", vecC, SIZE );
	// cleaning
	delete[] vecA;
	delete[] vecB;
	delete[] vecC;
	// done


    return 0;
}