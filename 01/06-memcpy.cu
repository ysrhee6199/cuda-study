
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void)
{
    const int SIZE = 8;
    const float a[SIZE] = {1.,2.,3.,4.,5.,6.,7.,8.};
    float b[SIZE] = {0.,0.,0.,0.,0.,0.,0.,0.};

    printf("a = {%f,%f,%f,%f,%f,%f,%f,%f} \n",a[0],a[1],a[2],a[3],a[4],a[5],a[6],a[7]);
    printf("b = {%f,%f,%f,%f,%f,%f,%f,%f} \n",b[0],b[1],b[2],b[3],b[4],b[5],b[6],b[7]);
    fflush(stdout);

    float* dev_a = nullptr;
    float* dev_b = nullptr;

    hipMalloc((void**)&dev_a, SIZE * sizeof(float));
    hipMalloc((void**)&dev_b, SIZE * sizeof(float));

    hipMemcpy(dev_a,a,SIZE*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,dev_a,SIZE*sizeof(float),hipMemcpyDeviceToDevice);
    hipMemcpy(b,dev_b,SIZE*sizeof(float),hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);

    hipDeviceSynchronize();

    printf("a = {%f,%f,%f,%f,%f,%f,%f,%f} \n",a[0],a[1],a[2],a[3],a[4],a[5],a[6],a[7]);
    printf("b = {%f,%f,%f,%f,%f,%f,%f,%f} \n",b[0],b[1],b[2],b[3],b[4],b[5],b[6],b[7]);
    fflush(stdout);

    return 0;

}