#include "hip/hip_runtime.h"
#include "./common.cpp"
//CUDA_CHECK_ERROR();
unsigned vecSize = 256 * 1024 * 1024;
float lerp_t = 0.234f;

__global__ void kernel_lerp(float* z, const float t, const float* x, const float* y, unsigned n)
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
    {
        //z[i] = (1.0F - t)*x[i] + t * y[i];
        z[i] = fmaf(t,y[i],fmaf(-t,x[i],x[i]));
    }
}

int main()
{
    float* vecX = new float[vecSize];
    float* vecY = new float[vecSize];
    float* vecZ = new float[vecSize];
	// set random data
	srand( 0 );
	setNormalizedRandomData( vecX, vecSize );
	setNormalizedRandomData( vecY, vecSize );

    float* dev_vecX = nullptr;
    float* dev_vecY = nullptr;
    float* dev_vecZ = nullptr;

    hipMalloc((void**)&dev_vecX, vecSize * sizeof(float));
    hipMalloc((void**)&dev_vecY, vecSize * sizeof(float));
    hipMalloc((void**)&dev_vecZ, vecSize * sizeof(float));
    CUDA_CHECK_ERROR();

    ELAPSED_TIME_BEGIN(1);
    hipMemcpy(dev_vecX,vecX,vecSize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(dev_vecY,vecY,vecSize*sizeof(float),hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR();

    // CUDA kernel launch
	dim3 dimBlock( 1024, 1, 1 );
	dim3 dimGrid( (vecSize + (dimBlock.x - 1)) / dimBlock.x, 1, 1 );

    ELAPSED_TIME_BEGIN(0);
    kernel_lerp<<<dimGrid,dimBlock>>>(dev_vecZ,lerp_t,dev_vecX,dev_vecY,vecSize);
    hipDeviceSynchronize();
    ELAPSED_TIME_END(0);
    CUDA_CHECK_ERROR();

    hipMemcpy(vecZ,dev_vecZ,vecSize*sizeof(float),hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR();
    ELAPSED_TIME_END(1);

    
    hipFree(dev_vecX);
    hipFree(dev_vecY);
    hipFree(dev_vecZ);
    CUDA_CHECK_ERROR();

// check the result
	float sumX = getSum( vecX, vecSize );
	float sumY = getSum( vecY, vecSize );
	float sumZ = getSum( vecZ, vecSize );
	float diff = fabsf( sumZ - (lerp_t * sumX + sumY) );
	printf("SIZE = %d\n", vecSize);
	printf("a    = %f\n", lerp_t);
	printf("sumX = %f\n", sumX);
	printf("sumY = %f\n", sumY);
	printf("sumZ = %f\n", sumZ);
	printf("diff(sumZ, a*sumX+sumY) =  %f\n", diff);
	printf("diff(sumZ, a*sumX+sumY)/SIZE =  %f\n", diff / vecSize);
	printVec( "vecX", vecX, vecSize );
	printVec( "vecY", vecY, vecSize );
	printVec( "vecZ", vecZ, vecSize );
	// cleaning
	delete[] vecX;
	delete[] vecY;
	delete[] vecZ;
	// done
	return 0;

    
}