// nvcc hello-pararllel.cu -gencode arch=compute_86,code=sm_86 -o a.out

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(void)
{
    printf("hello CUDA%d!\n",threadIdx.x);
}

int main(void)
{
    hello<<<1,8>>>();
    hipDeviceSynchronize();
    fflush(stdout);
    return 0;   
}