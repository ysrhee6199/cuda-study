#include "hip/hip_runtime.h"
#include "./common.cpp"

__global__ void add_kernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    //host side memory
    const int SIZE = 5;
    const int a[SIZE] = {1,2,3,4,5};
    const int b[SIZE] = {10,20,30,40,50};
    int c[SIZE] = {0};

    //device side data
    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;

    hipMalloc((void**)&dev_a,SIZE*sizeof(int));
    hipMalloc((void**)&dev_b,SIZE*sizeof(int));
    hipMalloc((void**)&dev_c,SIZE*sizeof(int));

    hipMemcpy(dev_a,a,SIZE*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,SIZE*sizeof(int),hipMemcpyHostToDevice);

    add_kernel<<<1,SIZE>>>(dev_c,dev_a,dev_b);
    hipDeviceSynchronize();
    hipError_t err = hipPeekAtLastError();
    if(hipSuccess != err)
    {
        exit(1);
    }
    else printf("CUDA Success \n");

    hipMemcpy(c,dev_c,SIZE*sizeof(int),hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    printf("{%d,%d,%d,%d,%d} + {%d,%d,%d,%d,%d} = {%d,%d,%d,%d,%d}\n ", a[0],a[1],a[2],a[3],a[4],b[0],b[1],b[2],b[3],b[4],c[0],c[1],c[2],c[3],c[4]);
    //error check
    hipError_t err = hipGetLastError();
    if(hipSuccess != err)
    {
        printf("CUDA:ERROR: cuda failure \"%s\"\n", hipGetErrorString(err));
        exit(1);
    }
    else
    {
        printf("CUDA: success\n");
    }   
   
    fflush(stdout);

    return 0;

}