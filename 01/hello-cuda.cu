
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(void)
{
    printf("hello CUDA%d!\n",threadIdx.x);
}

int main(void)
{
    hello<<<1,81>>>();
    fflush(stdout);
    return 0;   
}